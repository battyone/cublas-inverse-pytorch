#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_inverse_pytorch_device_test.h"

__global__ void printDevice(float** input){
    printf("%f \n", input[99][99]);
    return;
}

void printDeviceHost(float** input){
    printDevice<<<1,1>>>(input);
}
